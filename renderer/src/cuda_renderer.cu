#include "hip/hip_runtime.h"
/*
 * cuda_renderer.cu
 *
 *  Created on: Aug 2, 2021
 *      Author: Andre
 */

#include "cuda_renderer.h"
#include "cuda_vector.cuh"
#include "cuda_utils.cuh"
#include "cuda_scene.cuh"
#include <iostream>
#include <stdio.h>
#include <chrono>

#define RANDOM_NUMBERS_PER_PHOTON 6

#if USE_DOUBLE_PRECISION
#define FLOATS_IN_MEMORY_LINE 4
#else
#define FLOATS_IN_MEMORY_LINE 8
#endif /* USE_DOUBLE_PRECISION */

namespace cuda {

// Store symbols here so as to avoid long argument list
// in kernel calls
struct Constants {
    Float *image;
    int x_res;
    int y_res;
    int z_res;

	unsigned int *random_state;
    Scene *scene;
    Medium *medium;
    Float weight;
    Float3 blockL;
    Float3 blockR;

    int maxDepth;
    Float maxPathlength;
    bool useDirect;
    bool useAngularSampling;
    int numPhotons;
};

__constant__ Constants d_constants;

// Generate random unsigned int in [0, 2^24)
__host__ __device__ __inline__ unsigned int lcg(unsigned int &prev) {
  const unsigned int LCG_A = 1664525u;
  const unsigned int LCG_C = 1013904223u;
  prev = (LCG_A * prev + LCG_C);
  return prev & 0x00FFFFFF;
}

__host__ __device__ __inline__ Float float_rnd(unsigned int &rand) {
  return ((Float) rand / (Float) 0x01000000);
}

__device__ inline void init_rand(int idx) {
	d_constants.random_state[idx] += idx;
}
		
__device__ inline Float uniform_sample() {
    int idx = gridDim.x * blockDim.x * blockDim.y * blockIdx.y + blockDim.x * blockDim.y * blockIdx.x + blockDim.x * threadIdx.y + threadIdx.x;
	unsigned int curr = lcg(d_constants.random_state[idx]);
	d_constants.random_state[idx] = curr;
	Float rand = float_rnd(curr);
    return rand;
}

__device__ inline Float safeSqrt(Float x) {
    return x > FPCONST(0.0) ? sqrtf(x) : FPCONST(0.0);
}

__device__ inline void reflect(const TVector3<Float> &a, const TVector3<Float> &n,
                               TVector3<Float> &b) {
    b = -FPCONST(2.0)*dot(a, n)*n + a;
}

__device__ inline bool refract(const TVector3<Float> &a, const TVector3<Float> &n, Float eta, TVector3<Float> &b) {
    TVector3<Float> q = dot(a,n)*n;
    TVector3<Float> p = (a-q)/eta;

    if (p.length() > FPCONST(1.0)) {
        reflect(a, n, b);
        return false;
    } else {
        q.normalize();
        q *= safeSqrt(FPCONST(1.0) - p.lengthSquared());
        b = p + q;
        return true;
    }
}

__device__ inline Float fresnelDielectric(Float cosThetaI, Float cosThetaT, Float eta) {
	if (fabsf(eta - FPCONST(1.0)) < M_EPSILON * max(FPCONST(1.0), fabsf(eta))) {
		return FPCONST(0.0);
	} else {
		Float Rs = (cosThetaI - eta * cosThetaT) / (cosThetaI + eta * cosThetaT);
		Float Rp = (cosThetaT - eta * cosThetaI) / (cosThetaT + eta * cosThetaI);

		return FPCONST(0.5) * (Rs * Rs + Rp * Rp);
	}
}

__device__ inline void SmoothDielectric::sample(const TVector3<Float> &in, const TVector3<Float> &n,
				TVector3<Float> &out) const {
	if (fabsf(m_ior1 - m_ior2) < M_EPSILON) {
		// index matched
		out = in;
	} else {
		Float eta;
		if (dot(in, n) < -M_EPSILON) {
			// entering ior2 from ior1
			eta = m_ior2/m_ior1;
		}
		else {
			// entering ior1 from ior2
			eta = m_ior1/m_ior2;
		}

		TVector3<Float> outT;
		if (!refract(in, n, eta, outT)) {
			// TIR
			out = outT;
		} else {
			TVector3<Float> outR;
			reflect(in, n, outR);

			Float cosI = absDot(n, in), cosT = absDot(n, outT);
			Float fresnelR = fresnelDielectric(cosI, cosT, eta);

			// return either refracted or reflected direction based on the Fresnel term
			out = (uniform_sample() < fresnelR ? outR : outT);
		}
	}
}

// Sample random ray
__device__ bool area_textured_source::sample_ray(TVector3<Float> &pos, TVector3<Float> &dir,
                                              Float &total_distance) const{
    pos = *m_origin;

    // sample pixel position first
	int pixel = m_texture_sampler->sample(uniform_sample());
	int p[2];
	m_texture->ind2sub(pixel, p[0], p[1]);

	// Now find a random location on the pixel
	for (int iter = 1; iter < m_origin->dim; ++iter) {
		pos[iter] += - (*m_plane)[iter - 1] / FPCONST(2.0) +
            p[iter - 1] * (*m_pixelsize)[iter-1] + uniform_sample() * (*m_pixelsize)[iter - 1];
	}

	dir = *m_dir;

	//FIXME: Hack: Works only for m_dir = [-1 0 0]
	Float z   = uniform_sample()*(1-m_ct) + m_ct;
	Float zt  = sqrtf(FPCONST(1.0)-z*z);
	Float phi = uniform_sample()*2*M_PI;
	dir[0] = -z;
	dir[1] = zt*cosf(phi);
	dir[2] = zt*sinf(phi);
	
	return propagate_till_medium(pos, dir, total_distance);
}


__device__ inline Float getMoveStep(const Medium *medium) {
    return -medium->getMfp() * logf(uniform_sample());
}

__device__ Float henyey_greenstein::sample(const TVector2<Float> &in,
                        TVector2<Float> &out)  const {
    Float sampleVal = FPCONST(1.0) - FPCONST(2.0) * uniform_sample();

    Float theta;
    if (fabsf(m_g) < M_EPSILON) {
        theta = M_PI * sampleVal;
    } else {
        theta = FPCONST(2.0) * atanf((FPCONST(1.0) - m_g) / (FPCONST(1.0) + m_g)
                            * tanf(M_PI / FPCONST(2.0) * sampleVal));
    }
    Float cosTheta = cosf(theta);
    Float sinTheta = sinf(theta);

    TVector2<Float> axisY;
    axisY = TVector2<Float>(in.y, -in.x); // coordinate system

    out = sinTheta * axisY + cosTheta * in;
    return cosTheta;
}

__device__ Float henyey_greenstein::sample(const TVector3<Float> &in, TVector3<Float> &out) const {

    Float samplex = uniform_sample();
    Float sampley = uniform_sample();

    Float cosTheta;
    if (fabsf(m_g) < M_EPSILON) {
        cosTheta = 1 - 2 * samplex;
    } else {
        Float sqrTerm = (1 - m_g * m_g) / (1 - m_g + 2 * m_g * samplex);
        cosTheta = (1 + m_g * m_g - sqrTerm * sqrTerm) / (2 * m_g);
    }

    Float sinTheta = sqrtf(fmaxf(FPCONST(0.0), FPCONST(1.0) - cosTheta * cosTheta));
    Float phi = static_cast<Float>(FPCONST(2.0) * M_PI) * sampley;
    Float sinPhi, cosPhi;
    sinPhi = sinf(phi);
    cosPhi = cosf(phi);

    TVector3<Float> axisX, axisY;
    coordinateSystem(in, axisX, axisY);

    out = (sinTheta * cosPhi) * axisX + (sinTheta * sinPhi) * axisY + cosTheta * in;
    return cosTheta;
}

__device__ inline bool Camera::sample_position(TVector3<Float> &pos) const {
    pos = *m_origin;
    for (int iter = 1; iter < m_origin->dim; ++iter) {
        pos[iter] += - (*m_plane)[iter - 1] / FPCONST(2.0) + uniform_sample() * (*m_plane)[iter - 1];
    }
    return true;
}

__device__ inline void Scene::er_step(TVector3<Float> &p, TVector3<Float> &d, Float stepSize, Float scaling) const{
#ifndef OMEGA_TRACKING
    d += HALF * stepSize * dV(p, d, scaling);
    p +=        stepSize * d/m_us->RIF(p, scaling);
    d += HALF * stepSize * dV(p, d, scaling);
#else
    Float two = 2; // To avoid type conversion

    TVector3<Float> K1P = stepSize * dP(d);
    TVector3<Float> K1O = stepSize * dOmega(p, d);

    TVector3<Float> K2P = stepSize * dP(d + HALF*K1O);
    TVector3<Float> K2O = stepSize * dOmega(p + HALF*K1P, d + HALF*K1O);

    TVector3<Float> K3P = stepSize * dP(d + HALF*K2O);
    TVector3<Float> K3O = stepSize * dOmega(p + HALF*K2P, d + HALF*K2O);

    TVector3<Float> K4P = stepSize * dP(d + K3O);
    TVector3<Float> K4O = stepSize * dOmega(p + K3P, d + K3O);

    p = p + ONE_SIXTH * (K1P + two*K2P + two*K3P + K4P);
    d = d + ONE_SIXTH * (K1O + two*K2O + two*K3O + K4O);
#endif
}


__device__ inline bool inside_block(const TVector3<Float> &p) {
    float3 blockL = d_constants.blockL;
    float3 blockR = d_constants.blockR;

    return (p.x - blockL.x > -M_EPSILON)
        && (blockR.x - p.x > -M_EPSILON)
        && (p.y - blockL.y > -M_EPSILON)
        && (blockR.y - p.y > -M_EPSILON)
        && (p.z - blockL.z > -M_EPSILON)
        && (blockR.z - p.z > -M_EPSILON);
}

__device__ void Scene::traceTillBlock(TVector3<Float> &p, TVector3<Float> &d, Float dist, Float &disx, Float &disy, Float &totalOpticalDistance, Float scaling) const{
	TVector3<Float> oldp, oldd;

    Float distance = 0;
    long int maxsteps = dist/m_us->er_stepsize + 1, i, precision = m_us->getPrecision();

    Float current_stepsize = m_us->er_stepsize;

    for(i = 0; i < maxsteps; i++){
    	oldp = p;
    	oldd = d;

    	er_step(p, d, current_stepsize, scaling);

    	// check if we are at the intersection or crossing the sampled dist, then, estimate the distance and keep going more accurately towards the boundary or sampled dist
    	if(!inside_block(p) || (distance + current_stepsize) > dist){
    		precision--;
    		if(precision < 0)
    			break;
    		p = oldp;
    		d = oldd;
    		current_stepsize = current_stepsize / 10;
    		i  = 0;
    		maxsteps = 11;
    	}else{
    		distance += current_stepsize;
#if !USE_SIMPLIFIED_TIMING
    		totalOpticalDistance += current_stepsize * m_us->RIF(p, scaling);
#endif
    	}
    }

    disx = 0;
    disy = distance;
}

__device__ TVector3<Float> squareToUniformHemisphere(const TVector2<Float> &sample) {
	Float z = sample.y;
	Float r = sqrtf(FPCONST(1.0) - z*z);
	Float sinPhi, cosPhi;
	sincosf(FPCONST(2.0) * M_PI * sample.x, &sinPhi, &cosPhi);
	return TVector3<Float>(r * cosPhi, r * sinPhi, z);
}

__device__ void sampleRandomDirection(TVector3<Float> &randDirection){
	randDirection = squareToUniformHemisphere(TVector2<Float>(uniform_sample(), uniform_sample())); // this sampling is done in z=1 direction. need to compensate for it.
	Float temp = randDirection.x;
	randDirection.x =-randDirection.z; // compensating that the direction of photon propagation is -x
	randDirection.z = randDirection.y;
	randDirection.y = temp;
}

__device__ void Scene::addEnergyInParticle(const TVector3<Float> &p, const TVector3<Float> &d, Float distTravelled,
                                           int &depth, Float val, const Float &scaling) const {

	TVector3<Float> p1 = p;

	TVector3<Float> dirToSensor;

	if( (p.x-m_camera->get_origin().x) < 1e-4) // Hack to get rid of inf problems for direct connection
		return;

	sampleRandomDirection(dirToSensor); // Samples by assuming that the sensor is in +x direction.

#ifndef OMEGA_TRACKING
	dirToSensor *= getMediumIor(p1, scaling);
#endif

	Float distToSensor;
	if(!movePhotonTillSensor(p1, dirToSensor, distToSensor, distTravelled, scaling))
		return;

//if OMEGA_TRACKING
	dirToSensor.normalize();
//#endif

	TVector3<Float> refrDirToSensor = dirToSensor;
	Float fresnelWeight = FPCONST(1.0);
	Float ior = getMediumIor(p1, scaling);

	if (ior > FPCONST(1.0)) {
		refrDirToSensor.x = refrDirToSensor.x/ior;
		refrDirToSensor.normalize();
#ifndef USE_NO_FRESNEL
		fresnelWeight = (FPCONST(1.0) -
		fresnelDielectric(dirToSensor.x, refrDirToSensor.x,
			FPCONST(1.0) / ior))
			/ ior / ior;
#endif
	}
	Float foreshortening = dot(refrDirToSensor, m_camera->get_dir())/dot(dirToSensor, m_camera->get_dir());
	ASSERT(foreshortening >= FPCONST(0.0));

#if USE_SIMPLIFIED_TIMING
	Float totalOpticalDistance = (distTravelled + distToSensor) * m_ior;
#else
	Float totalOpticalDistance = distTravelled;
#endif

	Float distanceToSensor = 0;
	if(!m_camera->propagate_till_sensor(p1, refrDirToSensor, distanceToSensor))
		return;
	totalOpticalDistance += distanceToSensor;

    Medium *medium = d_constants.medium;

	Float totalPhotonValue = val*(2*M_PI)
			* expf(-medium->getSigmaT() * distToSensor)
			* medium->getPhaseFunction()->f(d/d.length(), dirToSensor) // FIXME: Should be refractive index
			* foreshortening
			* fresnelWeight;
	addEnergyToImage(p1, totalOpticalDistance, depth, totalPhotonValue);
}

__device__ bool Scene::movePhotonTillSensor(TVector3<Float> &p, TVector3<Float> &d, Float &distToSensor, Float &totalOpticalDistance,
                                            const Float& scaling) const {

	Float LargeDist = FPCONST(10000.0);

	Float disx, disy;
	TVector3<Float> d1, norm;
    TVector3<Float> blockL(d_constants.blockL.x, d_constants.blockL.y, d_constants.blockL.z);
    TVector3<Float> blockR(d_constants.blockR.x, d_constants.blockR.y, d_constants.blockR.z);
	traceTillBlock(p, d, LargeDist, disx, disy, totalOpticalDistance, scaling);
	distToSensor = disy;
	LargeDist -= disy;
	while(true){
		if(LargeDist < 0){
			return false;
		}
		int i;
		norm.zero();
		for (i = 0; i < p.dim; ++i) {
			if (fabsf(blockL[i] - p[i]) < 2*M_EPSILON) {
				norm[i] = -FPCONST(1.0);
				break;
			}
			else if (fabsf(blockR[i] - p[i]) < 2*M_EPSILON) {
				norm[i] = FPCONST(1.0);
				break;
			}
		}

		Float minDiff = M_MAX;
		Float minDir = FPCONST(0.0);
		TVector3<Float> normalt;
		normalt.zero();
		int chosenI = p.dim;
		for (i = 0; i < p.dim; ++i) {
			Float diff = fabsf(blockL[i] - p[i]);
			if (diff < minDiff) {
				minDiff = diff;
				chosenI = i;
				minDir = -FPCONST(1.0);
			}
			diff = fabsf(blockR[i] - p[i]);
			if (diff < minDiff) {
				minDiff = diff;
				chosenI = i;
				minDir = FPCONST(1.0);
			}
		}
		normalt[chosenI] = minDir;
		norm = normalt; // A HACK

        // check if we hit the sensor plane
		if(fabsf(m_camera->get_dir().x - norm.x) < M_EPSILON &&
				fabsf(m_camera->get_dir().y - norm.y) < M_EPSILON &&
				fabsf(m_camera->get_dir().z - norm.z) < M_EPSILON)
			return true;

		// if not, routine
        m_bsdf->sample(d, norm, d1);
		if (dot(d1, norm) < FPCONST(0.0)) {
			// re-enter the medium through reflection
			d = d1;
		} else {
			return false;
		}

    	traceTillBlock(p, d, LargeDist, disx, disy, totalOpticalDistance, scaling);
    	distToSensor += disy;
    	LargeDist -= disy;
	}

	return true;
}

__device__ inline void addPixel(int x, int y, int z, Float val) {
    Float *image = d_constants.image;
    int x_res = d_constants.x_res;
    int y_res = d_constants.y_res;
    int z_res = d_constants.z_res;

    if (x >= 0 && x < x_res && y >= 0 && y < y_res &&
        z >= 0 && z < z_res) {
        // atomicAdd is atomic within compute device.
        // For coherence with CPU/multiple GPUs, use atomicAdd_system
        atomicAdd(image + (z * x_res * y_res + y * x_res + x), val);
    }
}

__device__ void Scene::addEnergyToImage(const TVector3<Float> &p, Float pathlength, int &depth, Float val) const {
	Float x = dot(m_camera->get_horizontal(), p) - m_camera->get_origin().y;
	Float y = dot(m_camera->get_vertical(), p) - m_camera->get_origin().z;

	if (((m_camera->get_pathlength_range().x == -1) && (m_camera->get_pathlength_range().y == -1)) ||
		((pathlength > m_camera->get_pathlength_range().x) && (pathlength < m_camera->get_pathlength_range().y))) {
		x = (x / m_camera->get_plane().x + FPCONST(0.5)) * static_cast<Float>(d_constants.x_res);
		y = (y / m_camera->get_plane().y + FPCONST(0.5)) * static_cast<Float>(d_constants.y_res);

		int ix = static_cast<int>(floorf(x));
		int iy = static_cast<int>(floorf(y));

		int iz;
		if(m_camera->is_bounce_decomposition()){
			iz = depth;
		}
		else{
			if ((m_camera->get_pathlength_range().x == -1) && (m_camera->get_pathlength_range().y == -1)) {
				iz = 0;
			} else {
				Float z = pathlength - m_camera->get_pathlength_range().x;
				Float range = m_camera->get_pathlength_range().y - m_camera->get_pathlength_range().x;
				z = (z / range) * static_cast<Float>(d_constants.z_res);
				iz = static_cast<int>(floorf(z));
			}
		}
#if USE_PIXEL_SHARING
		Float fx = x - floorf(x);
		Float fy = y - floorf(y);

		addPixel(ix, iy, iz, val*(FPCONST(1.0) - fx)*(FPCONST(1.0) - fy));
		addPixel(ix + 1, iy, iz, val*fx*(FPCONST(1.0) - fy));
		addPixel(ix, iy + 1, iz, val*(FPCONST(1.0) - fx)*fy);
		addPixel(ix + 1, iy + 1, iz, val*fx*fy);
#else
		addPixel(ix, iy, iz, val);
#endif
    }
}

// Move photon and return true if still in medium, false otherwise
__device__ bool Scene::movePhoton(TVector3<Float> &p, TVector3<Float> &d, Float dist,
                                  Float &totalOpticalDistance, Float scaling) const{

	// Algorithm
	// 1. Move till you reach the boundary or till the distance is reached.
	// 2. If you reached the boundary, reflect with probability and keep progressing TODO: change to weight


	Float disx, disy;
	TVector3<Float> d1, norm;
	traceTillBlock(p, d, dist, disx, disy, totalOpticalDistance, scaling);

	dist -= disy;

    TVector3<Float> blockL(d_constants.blockL.x, d_constants.blockL.y, d_constants.blockL.z);
    TVector3<Float> blockR(d_constants.blockR.x, d_constants.blockR.y, d_constants.blockR.z);
	while(dist > M_EPSILON){
		int i;
		norm.zero();
		for (i = 0; i < p.dim; ++i) {
			if (fabsf(blockL[i] - p[i]) < M_EPSILON) {
				norm[i] = -FPCONST(1.0);
				break;
			}
			else if (fabsf(blockR[i] - p[i]) < M_EPSILON) {
				norm[i] = FPCONST(1.0);
				break;
			}
		}

		Float minDiff = M_MAX;
		Float minDir = FPCONST(0.0);
		TVector3<Float> normalt;
		normalt.zero();
		int chosenI = p.dim;
		for (i = 0; i < p.dim; ++i) {
			Float diff = fabsf(blockL[i] - p[i]);
			if (diff < minDiff) {
				minDiff = diff;
				chosenI = i;
				minDir = -FPCONST(1.0);
			}
			diff = fabsf(blockR[i] - p[i]);
			if (diff < minDiff) {
				minDiff = diff;
				chosenI = i;
				minDir = FPCONST(1.0);
			}
		}
		normalt[chosenI] = minDir;
		norm = normalt;

		/*
		 * TODO: I think that, because we always return to same medium (we ignore
		 * refraction), there is no need to adjust radiance by eta*eta.
		 */
		Float magnitude = d.length();
        m_bsdf->sample(d/magnitude, norm, d1);
        if (dot(d1, norm) < FPCONST(0.0)) {
			// re-enter the medium through reflection
			d = d1*magnitude;
		} else {
			return false;
		}

    	traceTillBlock(p, d, dist, disx, disy, totalOpticalDistance, scaling);
    	dist -= disy;
	}
	return true;
}

__device__ bool scatterOnce(TVector3<Float> &p, TVector3<Float> &d, Float &dist,
                            Float &totalOpticalDistance, const Float &scaling) {
    Medium *medium = d_constants.medium;
    Scene *scene = d_constants.scene;

	if ((medium->getAlbedo() > FPCONST(0.0)) && ((medium->getAlbedo() >= FPCONST(1.0)) || (uniform_sample() < medium->getAlbedo()))) {
		TVector3<Float> d1;
		Float magnitude = d.length();
		medium->getPhaseFunction()->sample(d/magnitude, d1);
		d = magnitude*d1;
		dist = getMoveStep(medium);
		return scene->movePhoton(p, d, dist, totalOpticalDistance, scaling);
	} else {
		dist = FPCONST(0.0);
		return false;
	}
}

__device__ void directTracing(const TVector3<Float> &p, const TVector3<Float> &d, const Float &scaling, Float &totalOpticalDistance) {

    const Camera &camera = d_constants.scene->getCamera();

	TVector3<Float> p1 = p;
	TVector3<Float> d1 = d;

	Float distToSensor;
	if(!d_constants.scene->movePhotonTillSensor(p1, d1, distToSensor, totalOpticalDistance, scaling))
		return;
	Float fresnelWeight = FPCONST(1.0);

#ifndef OMEGA_TRACKING
	d1.normalize();
#endif
	Float ior = d_constants.scene->getMediumIor(p1, scaling);
	TVector3<Float> refrDirToSensor = d1;

	if (ior > FPCONST(1.0)) {
		refrDirToSensor.x = refrDirToSensor.x/ior;
		refrDirToSensor.normalize();
#ifndef USE_NO_FRESNEL
		fresnelWeight = (FPCONST(1.0) -
		fresnelDielectric(d1.x, refrDirToSensor.x,
			FPCONST(1.0) / ior))
			/ ior / ior;
#endif
	}

	Float foreshortening = dot(refrDirToSensor, camera.get_dir())/dot(d1, camera.get_dir());
	ASSERT(foreshortening >= FPCONST(0.0));

#if USE_SIMPLIFIED_TIMING
	total_distance = (distToSensor) * ior;
#endif

	Float distanceToSensor = 0;
	if(!camera.propagate_till_sensor(p1, refrDirToSensor, distanceToSensor))
		return;
	totalOpticalDistance += distanceToSensor;

	Float totalPhotonValue = d_constants.weight
			* expf(-d_constants.medium->getSigmaT() * distToSensor)
			* fresnelWeight;
	int depth = 0;
	d_constants.scene->addEnergyToImage(p1, totalOpticalDistance, depth, totalPhotonValue);
}

__device__ void scatter(TVector3<Float> &p, TVector3<Float> &d, Float scaling, Float &totalOpticalDistance) {
    Scene *scene = d_constants.scene;
    Medium *medium = d_constants.medium;
	ASSERT(inside_block(p));

	if ((medium->getAlbedo() > FPCONST(0.0)) && ((medium->getAlbedo() >= FPCONST(1.0)) || (uniform_sample() < medium->getAlbedo()))) {
		TVector3<Float> pos(p), dir(d);

		Float dist = getMoveStep(medium);

		if (!scene->movePhoton(pos, dir, dist, totalOpticalDistance, scaling)) {
			return;
		}

		int depth = 1;
		Float totalDist = dist;
		while ((d_constants.maxDepth < 0 || depth <= d_constants.maxDepth) &&
				(d_constants.maxPathlength < 0 || totalDist <= d_constants.maxPathlength)) {
            ASSERT(d_constants.useAngularSampling);
			if(d_constants.useAngularSampling)
                scene->addEnergyInParticle(pos, dir, totalOpticalDistance, depth, d_constants.weight, scaling);
			if (!scatterOnce(pos, dir, dist, totalOpticalDistance, scaling)){
				break;
			}
#if USE_SIMPLIFIED_TIMING
			totalOpticalDistance += dist;
#endif
			++depth;
		}
	}
}

__global__ void renderPhotons() {
    TVector3<Float> pos;
    TVector3<Float> dir;
    Float total_distance = 0;
    Float scaling = 0;

    Scene scene = *d_constants.scene;

    int idx = gridDim.x * blockDim.x * blockDim.y * blockIdx.y + blockDim.x * blockDim.y * blockIdx.x + blockDim.x * threadIdx.y + threadIdx.x;

    if (idx < d_constants.numPhotons) {
	init_rand(idx);
        if (scene.genRay(pos, dir, total_distance)) {
#if USE_RIF_SOURCES
            scaling = 1.0f;
#else
            scaling = max(min(sinf(scene.getUSPhi_min() + scene.getUSPhi_range() * uniform_sample()), scene.getUSMaxScaling()), -scene.getUSMaxScaling());
#endif
#ifndef OMEGA_TRACKING
  	    dir *= scene.getMediumIor(pos, scaling);
#endif
            if (d_constants.useDirect)
                directTracing(pos, dir, scaling, total_distance); // Traces and adds direct energy, which is equal to weight * exp( -u_t * path_length);

            scatter(pos, dir, scaling, total_distance);
        }
    }
}

void CudaRenderer::renderImage(image::SmallImage& target, const med::Medium &medium, const scn::Scene<tvec::TVector3> &scene, int numPhotons) {

    setup(target, medium, scene, numPhotons);

    dim3 threadGrid(16, 16); // Arbitrary choice, total can go up to 1024 on most architectures, 2048 or 4096 on newer ones.
    int threadsPerBlock = threadGrid.x * threadGrid.y;
    int numBlocks = (numPhotons + threadsPerBlock - 1) / (threadsPerBlock);
    int width = 32; // Arbitrary as well

    // N + (W - 1) / W, to ensure we have enough threads as division rounds down
    dim3 blockGrid((numBlocks + width - 1) / width, width);
    CUDA_CALL(hipDeviceSynchronize());

    printf("rendering numblocks=%d, numPhotons=%d\n", numBlocks, numPhotons);
    renderPhotons<<<blockGrid,threadGrid>>>();
    CUDA_CALL(hipDeviceSynchronize());

    CUDA_CALL(hipMemcpy(image, cudaImage,
                         target.get_x_res()*target.get_y_res()*target.getZRes()*sizeof(Float),
                         hipMemcpyDeviceToHost));

    // Copy back to target.
    for (int x=0; x < target.get_x_res(); ++x) {
        for (int y=0; y < target.get_y_res(); ++y) {
            for (int z=0; z < target.getZRes(); ++z) {
                // Same calculation as addPixel (aka image.addEnergy)
                target.setPixel(x, y, z, image[z * target.get_x_res() * target.get_y_res() + y * target.get_x_res() + x]);
            }
        }
    }

    cleanup();
}

/* Allocates host and device data and sets up RNG. */
void CudaRenderer::setup(image::SmallImage& target, const med::Medium &medium, const scn::Scene<tvec::TVector3> &scene, int numPhotons) {
    /* Allocate host memory */
    image = new Float[target.get_x_res()*target.get_y_res()*target.getZRes()*sizeof(Float)];

    /* Allocate device memory*/
    CUDA_CALL(hipMalloc((void **)&cudaImage,
                         target.get_x_res()*target.get_y_res()*target.getZRes()*sizeof(Float)));
    CUDA_CALL(hipMemset(cudaImage, 0, target.get_x_res()*target.get_y_res()*target.getZRes()*sizeof(Float))); 
    Scene *cudaScene = Scene::from(scene);
    Medium *cudaMedium = Medium::from(medium);

    /* Setup hiprand state. */
	srand(time(0));

	int seed = rand();

    CUDA_CALL(hipMalloc((void **)&cudaRandomState, numPhotons * sizeof(unsigned int)));
	CUDA_CALL(hipMemset(cudaRandomState, seed, numPhotons * sizeof(unsigned int)));

    scn::block<tvec::TVector3> block = scene.getMediumBlock();

    /* Send in parameter pointers to device */
    Constants h_constants = {
        .image              = cudaImage,
        .x_res              = target.get_x_res(),
        .y_res              = target.get_y_res(),
        .z_res              = target.getZRes(),
		.random_state       = cudaRandomState,
        .scene              = cudaScene,
        .medium             = cudaMedium,
        .weight             = getWeight(medium, scene, numPhotons),
        .blockL             = make_float3(block.get_block_l().x, block.get_block_l().y, block.get_block_l().z),
        .blockR             = make_float3(block.get_block_r().x, block.get_block_r().y, block.get_block_r().z),
        .maxDepth           = maxDepth,
        .maxPathlength      = maxPathlength,
        .useDirect          = useDirect,
        .useAngularSampling = useAngularSampling,
        .numPhotons         = numPhotons
    };

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_constants), &h_constants, sizeof(Constants)));
}

void CudaRenderer::cleanup() {
    if (image) delete[] image;

	CUDA_CALL(hipFree(cudaImage));
	CUDA_CALL(hipFree(cudaRandomState));
}

CudaRenderer::~CudaRenderer() {}

}
